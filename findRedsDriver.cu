#include "hip/hip_runtime.h"
/****                                                                           
     File: findRedsDriver.cu
     By:  Ilya Nemtsov
     Compile: nvcc findRedsDriver.cu -o frgpu
     Run: ./frgpu

****/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUMPARTICLES 32768
#define NEIGHBORHOOD .05
#define THREADSPERBLOCK 128

     void initPos(float *);
     float findDistance(float *, int, int);
     __device__ float findDistanceGPU(float *, int, int);
     void dumpResults(int index[]);

     __global__ void findRedsGPU(float *p, int *numI);

     int main() {
      hipEvent_t start, stop;
      float time;

  float *pos, //host pos
        *dpos; //device pos
  int *numReds, //host numReds
      *dnumReds;//device numreds

  float psize = NUMPARTICLES * 4 * sizeof(float); //size of pos
  int nsize = NUMPARTICLES * sizeof(int); //size of numReds

  pos = (float *) malloc(psize); //allocate space for pos
  numReds = (int *) malloc(nsize); // allocate space for numreds

  initPos(pos); //initialize pos

  // your code to allocate device arrays for pos and numReds go here
  hipMalloc((void** )&dpos, psize);
  hipMalloc((void** )&dnumReds, nsize);

  // copy host pos to device pos
  hipMemcpy(dpos,pos,psize, hipMemcpyHostToDevice);

  // create timer events
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  /* invoke kernel findRedsGPU here */
  findRedsGPU<<<NUMPARTICLES/THREADSPERBLOCK,THREADSPERBLOCK>>>(dpos, dnumReds);

  hipDeviceSynchronize();

  // your code to copy results to numReds[] go here
  hipMemcpy(numReds,dnumReds,nsize,hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Elapsed time = %f\n", time);

  dumpResults(numReds);

  // clean up
  free(pos);
  hipFree(dpos);
  free(numReds);
  hipFree(dnumReds);

}

void initPos(float *p) {

  // your code for initializing pos goes here
  int i;
  int roll;
  for (i=0; i<NUMPARTICLES; i++) {
    p[i*4] = rand() / (float) RAND_MAX;
    p[i*4+1] = rand() / (float) RAND_MAX;
    p[i*4+2] = rand() / (float) RAND_MAX;
    roll = rand() % 3;
    if (roll == 0)
      p[i*4+3] = 0xff0000;
    else if (roll == 1)
      p[i*4+3] = 0x00ff00;
    else
      p[i*4+3] = 0x0000ff;
  }
}

__device__ float findDistanceGPU(float *p, int i, int j) {
  // your code for calculating distance for particle i and j
  float dx, dy, dz;

  dx = p[i*4] - p[j*4];
  dy = p[i*4+1] - p[j*4+1];
  dz = p[i*4+2] - p[j*4+2];

  return(sqrt(dx*dx + dy*dy + dz*dz));
}

__global__ void findRedsGPU(float *p, int *numI) {

  // your code for counting red particles goes here
  int j;
  float distance;

  int i = blockDim.x * blockIdx.x + threadIdx.x;

  numI[i]=0;
  for(j=0; j< NUMPARTICLES; j++){
    if(i!=j){
      distance = findDistanceGPU(p, i, j);
      if(distance < NEIGHBORHOOD && p[j*4+3] == 0xff0000){
        numI[i]++;
      }
    }
  }
}


void dumpResults(int index[]) {
  int i;
  FILE *fp;

  fp = fopen("./dump.out", "w");

  for (i=0; i<NUMPARTICLES; i++) {
    fprintf(fp, "%d %d\n", i, index[i]);
  }

  fclose(fp);
}
